#include "hip/hip_runtime.h"
#include "VX3_VoxelyzeKernel.cuh"
#include "VX3_MemoryCleaner.h"

/* Sub GPU Threads */
__global__ void gpu_update_force(VX3_Link** links, int num);
__global__ void gpu_update_voxel(VX3_Voxel* voxels, int num, double dt);
__global__ void gpu_update_temperature(VX3_Voxel* voxels, int num, double TempAmplitude, double TempPeriod, double currentTime);
__global__ void gpu_update_attach(VX3_Voxel** surface_voxels, int num, double watchDistance, VX3_VoxelyzeKernel* k);
/* Host methods */

VX3_VoxelyzeKernel::VX3_VoxelyzeKernel(CVX_Sim* In) {

    voxSize = In->Vx.voxSize;
    
    num_d_voxelMats = In->Vx.voxelMats.size();
    VcudaMalloc((void **)&d_voxelMats, num_d_voxelMats * sizeof(VX3_MaterialVoxel));
    {
        int i=0;
        for (auto mat: In->Vx.voxelMats) {
            VX3_MaterialVoxel tmp_voxelMat( mat, this );
            VcudaMemcpy( d_voxelMats+i, &tmp_voxelMat, sizeof(VX3_MaterialVoxel), VcudaMemcpyHostToDevice );
            h_voxelMats.push_back(mat);
            i++;
        }
    }

    num_d_linkMats = In->Vx.linkMats.size();
    VcudaMalloc( (void **)&d_linkMats, num_d_linkMats * sizeof(VX3_MaterialLink));
    {
        int i = 0;
        std::vector<VX3_MaterialLink*> tmp_v_linkMats;
        for (CVX_MaterialLink* mat:In->Vx.linkMats) {
            // printf("mat->vox1Mat %p, mat->vox2Mat %p.\n", mat->vox1Mat, mat->vox2Mat);
            VX3_MaterialLink tmp_linkMat( mat, this );
            VcudaMemcpy( d_linkMats+i, &tmp_linkMat, sizeof(VX3_MaterialLink), VcudaMemcpyHostToDevice );
            tmp_v_linkMats.push_back(d_linkMats+i);
            h_linkMats.push_back( mat );
            i++;
        }
        hd_v_linkMats = VX3_hdVector<VX3_MaterialLink*>(tmp_v_linkMats);
    }

    num_d_voxels = In->Vx.voxelsList.size();
    VcudaMalloc( (void **)&d_voxels, num_d_voxels * sizeof(VX3_Voxel));
    for (int i=0;i<num_d_voxels;i++) {
        h_voxels.push_back( In->Vx.voxelsList[i] );
    }

    num_d_links = In->Vx.linksList.size();
    std::vector<VX3_Link*> tmp_v_links;
    VcudaMalloc( (void **)&d_links, num_d_links * sizeof(VX3_Link));
    VX3_Link* tmp_link_cache = (VX3_Link*) malloc(num_d_links * sizeof(VX3_Link));
    for (int i=0;i<num_d_links;i++) {
        VX3_Link tmp_link( In->Vx.linksList[i], this );
        memcpy(tmp_link_cache+i, &tmp_link, sizeof(VX3_Link));
        tmp_v_links.push_back(d_links+i); //not copied yet, but still ok to get the address
        h_links.push_back( In->Vx.linksList[i] );
    }
    VcudaMemcpy( d_links, tmp_link_cache, num_d_links * sizeof(VX3_Link), VcudaMemcpyHostToDevice );
    hd_v_links = VX3_hdVector<VX3_Link*>(tmp_v_links);

    for (int i=0;i<num_d_voxels;i++) {
        //set values for GPU memory space
        VX3_Voxel tmp_voxel(In->Vx.voxelsList[i], this);
        VcudaMemcpy(d_voxels+i, &tmp_voxel, sizeof(VX3_Voxel), VcudaMemcpyHostToDevice);
    }

    //Not all data is in Vx, here are others:
    DtFrac = In->DtFrac;
    StopConditionType = In->StopConditionType;
    StopConditionValue = In->StopConditionValue;
    TempEnabled = In->pEnv->TempEnabled;
    VaryTempEnabled = In->pEnv->VaryTempEnabled;
    TempBase = In->pEnv->TempBase;
    TempAmplitude = In->pEnv->TempAmplitude;
    TempPeriod = In->pEnv->TempPeriod;
    // currentTemperature = TempBase + TempAmplitude;

    d_surface_voxels = NULL;
}

void VX3_VoxelyzeKernel::cleanup() {
    //The reason not use ~VX3_VoxelyzeKernel is that will be automatically call multiple times after we use memcpy to clone objects.
    MycudaFree(d_linkMats);
    MycudaFree(d_voxels);
    MycudaFree(d_links);
    MycudaFree(d_collisionsStale);
    if (d_surface_voxels) {
        MycudaFree(d_surface_voxels); //can __device__ malloc pointer be freed by hipFree in __host__??
    }
    // MycudaFree(d_collisions);
}

/* Cuda methods : cannot use any CVX_xxx, and no std::, no boost::, and no filesystem. */

__device__ void VX3_VoxelyzeKernel::syncVectors() {
    d_v_linkMats.clear();
    for (int i=0;i<hd_v_linkMats.size();i++) {
        d_v_linkMats.push_back(hd_v_linkMats[i]);
    }

    d_v_links.clear();
    for (int i=0;i<hd_v_links.size();i++) {
        d_v_links.push_back(hd_v_links[i]);
    }

    for (int i=0;i<num_d_voxelMats;i++) {
        d_voxelMats[i].syncVectors();
    }
    
    for (int i=0;i<num_d_linkMats;i++) {
        d_linkMats[i].syncVectors();
    }
}
__device__ bool VX3_VoxelyzeKernel::StopConditionMet(void) //have we met the stop condition yet?
{
    if (StopConditionType!=SC_MAX_SIM_TIME) {
        printf(COLORCODE_BOLD_RED "StopConditionType: %d. Type of stop condition no supported for now.\n" COLORCODE_RESET, StopConditionType);
        return true;
    }
    if (forceExit) return true;
    return currentTime > StopConditionValue ? true : false;
}

__device__ double VX3_VoxelyzeKernel::recommendedTimeStep() {
    //find the largest natural frequency (sqrt(k/m)) that anything in the simulation will experience, then multiply by 2*pi and invert to get the optimally largest timestep that should retain stability
	double MaxFreq2 = 0.0f; //maximum frequency in the simulation in rad/sec
    for (int i=0;i<num_d_links;i++) {
        VX3_Link* pL = d_links+i;
		//axial
		double m1 = pL->pVNeg->mat->mass(),  m2 = pL->pVPos->mat->mass();
		double thisMaxFreq2 = pL->axialStiffness()/(m1<m2?m1:m2);
		if (thisMaxFreq2 > MaxFreq2) MaxFreq2 = thisMaxFreq2;
		//rotational will always be less than or equal
	}
	if (MaxFreq2 <= 0.0f){ //didn't find anything (i.e no links) check for individual voxelss
		for (int i=0;i<num_d_voxels;i++){ //for each link
			double thisMaxFreq2 = d_voxels[i].mat->youngsModulus() * d_voxels[i].mat->nomSize / d_voxels[i].mat->mass(); 
			if (thisMaxFreq2 > MaxFreq2) MaxFreq2 = thisMaxFreq2;
		}
	}
	if (MaxFreq2 <= 0.0f) return 0.0f;
	else return 1.0f/(6.283185f*sqrt(MaxFreq2)); //the optimal timestep is to advance one radian of the highest natural frequency
}

__device__ void VX3_VoxelyzeKernel::updateTemperature() {
    //updates the temperatures For Actuation!
    // different temperatures in different objs are not support for now.
    if (VaryTempEnabled){
		if (TempPeriod > 0) {
            int blockSize = 512;
            int minGridSize;
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_temperature, 0, num_d_voxels); //Dynamically calculate blockSize
            int gridSize_voxels = (num_d_voxels + blockSize - 1) / blockSize; 
            int blockSize_voxels = num_d_voxels<blockSize ? num_d_voxels : blockSize;
            gpu_update_temperature<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, num_d_voxels, TempAmplitude, TempPeriod, currentTime);
            CUDA_CHECK_AFTER_CALL();
            hipDeviceSynchronize();        
        }
	}
}

__device__ bool VX3_VoxelyzeKernel::doTimeStep(float dt) {
    updateTemperature();
    CurStepCount++;
	if (dt==0) return true;
	else if (dt<0) {
        if (!OptimalDt) {
            OptimalDt = recommendedTimeStep();
        }
        if (OptimalDt<1e-10) {
            CUDA_DEBUG_LINE("recommendedTimeStep is zero.");
            return false;
        }
        dt = DtFrac*OptimalDt;
    }
    bool Diverged = false;

    int blockSize;
    int minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_force, 0, d_v_links.size()); //Dynamically calculate blockSize
    int gridSize_links = (d_v_links.size() + blockSize - 1) / blockSize; 
    int blockSize_links = d_v_links.size()<blockSize ? d_v_links.size() : blockSize;
    // printf("gpu_update_force<<<%d,%d>>>(...,%d);\n", gridSize_links, blockSize_links, d_v_links.size());
    gpu_update_force<<<gridSize_links, blockSize_links>>>(&d_v_links[0], d_v_links.size());
    CUDA_CHECK_AFTER_CALL();
    hipDeviceSynchronize();

    for (int i = 0; i<d_v_links.size(); i++){
        if (d_v_links[i]->axialStrain() > 100){
            CUDA_DEBUG_LINE("Diverged.");
            Diverged = true; //catch divergent condition! (if any thread sets true we will fail, so don't need mutex...
        }
    }
    if (Diverged) return false;

    if (enableAttach) updateAttach();

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_voxel, 0, num_d_voxels); //Dynamically calculate blockSize
    int gridSize_voxels = (num_d_voxels + blockSize - 1) / blockSize; 
    int blockSize_voxels = num_d_voxels<blockSize ? num_d_voxels : blockSize;
    gpu_update_voxel<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, num_d_voxels, dt);
    CUDA_CHECK_AFTER_CALL();
    hipDeviceSynchronize();

    currentTime += dt;
    return true;
}

__device__ void VX3_VoxelyzeKernel::updateAttach()
{
    //for each surface voxel pair, check distance < watchDistance, make a new link between these two voxels, updateSurface().
    int blockSize = 32;
    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid((num_d_surface_voxels + dimBlock.x - 1) / dimBlock.x, (num_d_surface_voxels + dimBlock.y - 1) / dimBlock.y);
    gpu_update_attach<<<dimGrid, dimBlock>>>(d_surface_voxels, num_d_surface_voxels, watchDistance, this); //invoke two dimensional gpu threads 'CUDA C++ Programming Guide', Nov 2019, P52.
    CUDA_CHECK_AFTER_CALL();
}


__device__ void VX3_VoxelyzeKernel::updateCurrentCenterOfMass() {
	double TotalMass = 0;
	VX3_Vec3D<> Sum(0,0,0);
	for (int i=0; i<num_d_voxels; i++){
        double ThisMass = d_voxels[i].material()->mass();
		Sum += d_voxels[i].position()*ThisMass;
        TotalMass += ThisMass;
	}

	currentCenterOfMass = Sum/TotalMass;
}

__device__ void VX3_VoxelyzeKernel::regenerateSurfaceVoxels() {
    // regenerate d_surface_voxels
    if (d_surface_voxels) {
        delete d_surface_voxels;
        d_surface_voxels = NULL;
    }
    VX3_dVector<VX3_Voxel*> tmp;
    for (int i=0;i<num_d_voxels;i++) {
        if (d_voxels[i].isSurface()) {
            tmp.push_back(&d_voxels[i]);
        }
    }
    num_d_surface_voxels = tmp.size();
    d_surface_voxels = (VX3_Voxel **)malloc( num_d_surface_voxels * sizeof(VX3_Voxel) );
    for (int i=0;i<num_d_surface_voxels;i++) {
        d_surface_voxels[i] = tmp[i];
    }
}

__device__ VX3_MaterialLink* VX3_VoxelyzeKernel::combinedMaterial(VX3_MaterialVoxel* mat1, VX3_MaterialVoxel* mat2) 
{
    for (int i=0;i<d_v_linkMats.size();i++) {
        VX3_MaterialLink* thisMat = d_v_linkMats[i];
		if ((thisMat->vox1Mat == mat1 && thisMat->vox2Mat == mat2) || (thisMat->vox1Mat == mat2 && thisMat->vox2Mat == mat1))
			return thisMat; //already exist
    }
    
    VX3_MaterialLink* newMat = new VX3_MaterialLink(mat1, mat2); //where to free this?
    d_v_linkMats.push_back(newMat);
	mat1->d_dependentMaterials.push_back(newMat);
	mat2->d_dependentMaterials.push_back(newMat);

	return newMat;
}


/* Sub GPU Threads */
__global__ void gpu_update_force(VX3_Link** links, int num) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < num) {
        VX3_Link* t = links[gindex];
        t->updateForces();
        if (t->axialStrain() > 100) { printf("ERROR: Diverged."); }
    }
}
__global__ void gpu_update_voxel(VX3_Voxel* voxels, int num, double dt) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x; 
    if (gindex < num) {
        VX3_Voxel* t = &voxels[gindex];
        t->timeStep(dt);
    }
}

__global__ void gpu_update_temperature(VX3_Voxel* voxels, int num, double TempAmplitude, double TempPeriod, double currentTime) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x; 
    if (gindex < num) {
    //vfloat tmp = pEnv->GetTempAmplitude() * sin(2*3.1415926f*(CurTime/pEnv->GetTempPeriod() + pV->phaseOffset)) - pEnv->GetTempBase();
        VX3_Voxel* t = &voxels[gindex];
        double currentTemperature = TempAmplitude*sin(2*3.1415926f*(currentTime/TempPeriod + t->phaseOffset));	//update the global temperature
        t->setTemperature(currentTemperature);
        // t->setTemperature(0.0f);
    }
}
__global__ void gpu_update_attach(VX3_Voxel** surface_voxels, int num, double watchDistance, VX3_VoxelyzeKernel* k) {
    int first = threadIdx.x + blockIdx.x * blockDim.x; 
    int second = threadIdx.y + blockIdx.y * blockDim.y; 
    if (first<num && second<first) {
        VX3_Voxel* voxel1 = surface_voxels[first];
        VX3_Voxel* voxel2 = surface_voxels[second];
        double diffx = voxel1->pos.x - voxel2->pos.x;
        double diffy = voxel1->pos.y - voxel2->pos.y;
        double diffz = voxel1->pos.z - voxel2->pos.z;
        if (diffx>watchDistance || diffx<-watchDistance) return;
        if (diffy>watchDistance || diffy<-watchDistance) return;
        if (diffz>watchDistance || diffz<-watchDistance) return;
        //to exclude voxels already have link between them.
        for (int i=0;i<6;i++) {
            if (voxel1->links[i]) {
                if (voxel1->links[i]->pVNeg == voxel2 || voxel1->links[i]->pVPos == voxel2) return;
            }
        }
        //create a link between voxel1 and voxel2 (orientation matters?)
        VX3_MaterialLink* mat = k->combinedMaterial(voxel1->material(), voxel2->material());
		VX3_Link* pL = new VX3_Link(voxel1, voxel2, mat); //make the new link (change to both materials, etc.
    
        k->d_v_links.push_back(pL);							//add to the list

        printf("hmmm.... %p %p distance=> %f %f %f\n", voxel1, voxel2, diffx, diffy, diffz);
    }
}