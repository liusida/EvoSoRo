#include "hip/hip_runtime.h"
#include <boost/algorithm/string/case_conv.hpp>
#include "VX3/VX3_SimulationManager.cuh"
#include "VX3_VoxelyzeKernel.h"
#include "VX_Sim.h"


__global__ void CUDA_Simulation(VX3_VoxelyzeKernel *d_voxelyze_3, int num_tasks) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<num_tasks) {
        VX3_VoxelyzeKernel *d_v3 = &d_voxelyze_3[i];
        d_v3->syncVectors(); //Everytime we pass a class with VX3_vectors in it, we should sync hd_vector to d_vector first.
        printf(COLORCODE_GREEN "Simulation %d runs. voxSize %f. \t" COLORCODE_RESET, i, d_v3->voxSize);
        // for (int j=0;j<1000000;j++) { //Maximum Steps 1000000
        //     if (d_v3->StopConditionMet()) break;
        //     // if (j%1000==0) {
        //     //     printf("----> [Task %d] doTimeStep %d, Current Time (in sec) %f \t", i, j, d_v3->currentTime);
        //     //     d_v3->updateCurrentCenterOfMass();
        //     //     printf("Current Location (in meter): %f %f %f\n", d_v3->currentCenterOfMass.x, d_v3->currentCenterOfMass.y, d_v3->currentCenterOfMass.z);
        //     // }
        //     if (!d_v3->doTimeStep()) {
        //         printf(COLORCODE_BOLD_RED "\nSimulation %d Diverged.\n" COLORCODE_RESET, i);
        //         break;
        //     }
        //     // if (j% 1000==0)
        //     //     printf("Time: %f, pos[0]: %f %f %f\n", d_v3->currentTime, d_v3->d_voxels[0].pos.x, d_v3->d_voxels[0].pos.y, d_v3->d_voxels[0].pos.z);

        // }
        // d_v3->updateCurrentCenterOfMass();
        printf(COLORCODE_BLUE "Simulation %d ends.\t" COLORCODE_RESET, i);
    }
}

VX3_SimulationManager::VX3_SimulationManager(fs::path input, fs::path output) : 
d_voxelyze_3(NULL), input_directory(input), output_file(output) {
    hipGetDeviceCount(&num_of_devices);
    streams.resize(num_of_devices);
    for (int i=0;i<num_of_devices;i++) {
        hipStreamCreate(&streams[i]);
    }
}
VX3_SimulationManager::~VX3_SimulationManager() {
    for (auto stream : streams) {
        hipStreamDestroy(stream);
    }
    VcudaFree(d_voxelyze_3);
}

void VX3_SimulationManager::start() {
    splitIntoSubBatches();
    int i=0;

    for (auto &files : sub_batches) {
        hipSetDevice(i);
        printf("=====%ld====\n", files.size());
        readVXA(files, i);
        startKernel(files.size(), i);
        i++;
    }
    hipDeviceSynchronize();
}

void VX3_SimulationManager::readVXA(std::vector<fs::path> files, int batch_index) {
    std::vector<std::string> filenames;
    int batch_size = files.size();
    
    VcudaMalloc((void**)&d_voxelyze_3, batch_size * sizeof(VX3_VoxelyzeKernel));
    
    int i = 0;
    for (auto &file : files ) {
        
        CVX_Environment MainEnv;
        CVX_Sim MainSim;
        CVX_Object MainObj;
        MainEnv.pObj = &MainObj; //connect environment to object
        MainSim.pEnv = &MainEnv; //connect Simulation to envirnment
        MainSim.LoadVXAFile(file.string());
        filenames.push_back(file.string());
        std::string err_string; //need to link this up to get info back...
        if (!MainSim.Import(NULL, NULL, &err_string)){
            std::cout<<err_string;
        }
        
        VX3_VoxelyzeKernel h_d_tmp(&MainSim.Vx, streams[batch_index]);
        h_d_tmp.DtFrac = MainSim.DtFrac;
        h_d_tmp.StopConditionType = MainSim.StopConditionType;
        h_d_tmp.StopConditionValue = MainSim.StopConditionValue;
        h_d_tmp.TempEnabled = MainSim.pEnv->TempEnabled;
        h_d_tmp.VaryTempEnabled = MainSim.pEnv->VaryTempEnabled;
        h_d_tmp.TempBase = MainSim.pEnv->TempBase;
        h_d_tmp.TempAmplitude = MainSim.pEnv->TempAmplitude;
        h_d_tmp.TempPeriod = MainSim.pEnv->TempPeriod;
        h_d_tmp.currentTemperature = h_d_tmp.TempBase + h_d_tmp.TempAmplitude;
        
        VcudaMemcpyAsync(d_voxelyze_3 + i, &h_d_tmp, sizeof(VX3_VoxelyzeKernel), VcudaMemcpyHostToDevice, streams[batch_index]);
        
        i++;
    }
}

void VX3_SimulationManager::splitIntoSubBatches() { //Sub-batches are for Multiple GPUs on one node.
    int i=0;
    sub_batches.clear(); //TODO: check if we can clear a vector of vector like this. memory leak?
    sub_batches.resize(num_of_devices);
    for (auto & file : fs::directory_iterator( input_directory )) {
        if (boost::algorithm::to_lower_copy(file.path().extension().string()) == ".vxa") {
            int iGPU = (i%num_of_devices);
            sub_batches[iGPU].push_back( file.path() );
            i++;
        }
    }
}

void VX3_SimulationManager::startKernel(int num_tasks, int batch_index) {
    int threadsPerBlock = 512;
    int numBlocks = (num_tasks + threadsPerBlock - 1) / threadsPerBlock;
    if (numBlocks == 1)
        threadsPerBlock = num_tasks;
    CUDA_Simulation<<<numBlocks,threadsPerBlock,0,streams[batch_index]>>>(d_voxelyze_3, num_tasks);
}

void VX3_SimulationManager::writeResults(int num_tasks) {
    // double final_z = 0.0;
    // VX3_VoxelyzeKernel* result_voxelyze_kernel = (VX3_VoxelyzeKernel *)malloc(num_tasks * sizeof(VX3_VoxelyzeKernel));
    
    // VcudaMemcpyAsync( result_voxelyze_kernel, d_voxelyze_3, num_tasks * sizeof(VX3_VoxelyzeKernel), VcudaMemcpyDeviceToHost, streams[num_tasks] );
    
    // printf("\n====[RESULTS for ]====\n");
    // std::vector< std::pair<double, int> > normAbsoluteDisplacement;
    // for (int i=0;i<num_tasks;i++) {
    //     double x = result_voxelyze_kernel[i].currentCenterOfMass.x;
    //     double y = result_voxelyze_kernel[i].currentCenterOfMass.y;
    //     double z = result_voxelyze_kernel[i].currentCenterOfMass.y;
    //     double v = result_voxelyze_kernel[i].voxSize;
    //     x = x/v; y = y/v; z = z/v;
    //     double dist = sqrt(x*x + y*y + z*z);
    //     normAbsoluteDisplacement.push_back( std::make_pair(dist,i) );
    // }
    // std::sort(normAbsoluteDisplacement.begin(), normAbsoluteDisplacement.end());
    // std::reverse(normAbsoluteDisplacement.begin(), normAbsoluteDisplacement.end());
    // pt::ptree xml_tree;
    // // xml_tree.put("voxelyzeManager.batchName", batchFolder.filename());
    // for (auto p : normAbsoluteDisplacement) {
    //     pt::ptree task;
    //     task.put("normAbsoluteDisplacement", p.first);
    //     task.put("taskId", p.second);
    //     // task.put("VXAFilename", filenames[p.second]);
    //     task.put("AbsoluteDistanceInMeter.x", result_voxelyze_kernel[p.second].currentCenterOfMass.x);
    //     task.put("AbsoluteDistanceInMeter.y", result_voxelyze_kernel[p.second].currentCenterOfMass.y);
    //     task.put("AbsoluteDistanceInMeter.z", result_voxelyze_kernel[p.second].currentCenterOfMass.z);
    //     task.put("VoxelSizeInMeter", result_voxelyze_kernel[p.second].voxSize);
    //     xml_tree.add_child("voxelyzeManager.Report", task);
    // }
    // pt::write_xml(output_file.string(), xml_tree, \
    //                     std::locale(), pt::xml_writer_make_settings<std::string>('\t', 1));
    // printf("Best distance of this generation is %f (x voxelSize).\n", normAbsoluteDisplacement[0].first);
    // printf("A detailed report.xml has been produced in the batch folder.\n");

    // delete result_voxelyze_kernel;
}
